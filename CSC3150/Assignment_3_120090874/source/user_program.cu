﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//test case 1
__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size) {

  for (int i = 0; i < input_size; i++){
    vm_write(vm, i, input[i]);
  }           

  for (int i = input_size - 1; i >= input_size - 32769; i--)
    int value = vm_read(vm, i); 

  vm_snapshot(vm, results, 0, input_size);
}

//test case 2
// __device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results, int input_size) {
// //   write the data.bin to the VM starting from address 32*1024
//   for (int i = 0; i < input_size; i++)
//         vm_write(vm, 32*1024+i, input[i]);               
// //   write some data (32KB-32B) to the VM starting from 0
//   for (int i = 0; i < 32*1023; i++)
//         vm_write(vm, i, input[i+32*1024]);               
// //   readout VM[32K, 160K] and output to snapshot.bin, which // should be the same with data.bin
//   vm_snapshot(vm, results, 32*1024, input_size);            
// }